#include "hip/hip_runtime.h"
#include "tak/tak.hpp"
#include "tak/ptn.hpp"
#include "tak/tps.hpp"
#include "hipcub/hipcub.hpp"
#include <chrono>

struct Eval {
  using Score = int32_t;

  enum S : Score {
    MIN = -(1<<30),
    MAX = (1<<30),
    LOSS = -(1<<29),
    WIN = 1<<29,
  };

  // Evaluates the strength of one player
  template<uint8_t SIZE>
  CUDA_CALLABLE static Score eval_player(const Board<SIZE>& state, uint8_t player) {
    int top_flats = 0;
    int adj_flats = 0;
    int flats = 0;
    int caps = 0;
    int influence = 0;
    int captured = 0;
    int captured_penalty = 0;
    for(int i = 0; i < SIZE*SIZE; i++) {
      Stack s = state.board[i];
      int cap_this_stack = 0;
      if(s.height && s.top == Piece::FLAT && s.owner() == player) {
        top_flats++;
        uint8_t o = i+Move<SIZE>::Dir::NORTH;
        if(o < SIZE*SIZE && state.board[o].height && state.board[o].owner() == player && state.board[o].top == Piece::FLAT) {
          adj_flats++;
        }
        o = i+Move<SIZE>::Dir::SOUTH;
        if(o < SIZE*SIZE && state.board[o].height && state.board[o].owner() == player && state.board[o].top == Piece::FLAT) {
          adj_flats++;
        }
        o = i+Move<SIZE>::Dir::EAST;
        if(o/SIZE == i/SIZE && state.board[o].height && state.board[o].owner() == player && state.board[o].top == Piece::FLAT) {
          adj_flats++;
        }
        o = i+Move<SIZE>::Dir::WEST;
        if(o/SIZE == i/SIZE && state.board[o].height && state.board[o].owner() == player && state.board[o].top == Piece::FLAT) {
          adj_flats++;
        }
        //influence += (0x7F&map.left[i]) + (0x7F&map.right[i]) + (0x7F&map.up[i]) + (0x7F&map.down[i]);
        uint64_t owners = state.board[i].owners;
        for(int i = 1; i < state.board[i].height; i++) {
          owners >>= 1;
          if((owners&1) == player) {
            flats += 1;
          } else {
            captured += 1;
            cap_this_stack += 1;
          }
        }
        if(cap_this_stack >= 3) {
          captured_penalty += cap_this_stack*cap_this_stack;
        }
      } else if(s.height && s.top == Piece::CAP && s.owner() == player) {
        caps++;
      }

      int adj_ally = 0;
      int adj_enemy = 0;
      uint8_t o = i+Move<SIZE>::Dir::NORTH;
      if(o < SIZE*SIZE && state.board[o].height) {
        if(state.board[o].owner() == player) {
          adj_ally++;
        } else {
          adj_enemy++;
        }
      }
      o = i+Move<SIZE>::Dir::SOUTH;
      if(o < SIZE*SIZE && state.board[o].height) {
        if(state.board[o].owner() == player) {
          adj_ally++;
        } else {
          adj_enemy++;
        }
      }
      o = i+Move<SIZE>::Dir::EAST;
      if(o/SIZE == i/SIZE && state.board[o].height) {
        if(state.board[o].owner() == player) {
          adj_ally++;
        } else {
          adj_enemy++;
        }
      }
      o = i+Move<SIZE>::Dir::WEST;
      if(o/SIZE == i/SIZE && state.board[o].height) {
        if(state.board[o].owner() == player) {
          adj_ally++;
        } else {
          adj_enemy++;
        }
      }

      influence += adj_ally-adj_enemy;
    }

    return influence*25 + (top_flats+adj_flats/2)*400 + flats*100 + caps*50 - captured_penalty*100;
  }

  template<uint8_t SIZE>
  CUDA_CALLABLE static Score eval(const Board<SIZE>& state, uint8_t player) {
    return eval_player(state, player) - eval_player(state, !player);
  }
};

const int BLOCK_SIZE = 128;
const int NUM_BLOCKS = 32;

template<uint8_t SIZE>
__global__ void eval_parallel(uint8_t player, int num_moves, Board<SIZE>* board, Move<SIZE>* moves, int* score) {
  //int this_score = Eval::MIN;
  for(int idx = blockIdx.x*blockDim.x + threadIdx.x; idx < num_moves; idx += gridDim.x*blockDim.x) {
    Board<SIZE> b = *board;
    b.execute(moves[idx]);
    //this_score = max(this_score, Eval::eval(b, player));
    *score = Eval::eval(b,player);
  }

  //int max_score = hipcub::BlockReduce<int, BLOCK_SIZE>().Reduce(this_score, hipcub::Max());

  // Only one thread in the block needs to write output
  //if(threadIdx.x == 0) {
    //*score = max_score;
  //}
}


//Macro for checking cuda errors following a cuda launch or api call
#define cudaCheckError() { \
hipError_t e=hipGetLastError(); \
  if(e!=hipSuccess) { \
    printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e)); \
    exit(0); \
  } \
}

int main() {
  using namespace std::chrono;

  Board<5> host_board;

  // Read in board from tps
  std::string board_tps;
  std::getline(std::cin, board_tps);
  tps::from_str(board_tps, host_board);
  std::cout << tps::to_str(host_board) << std::endl;

  std::vector<Move<5>> host_moves;
  std::vector<Move<5>> moves;

  typename Board<5>::Map map(host_board);
  host_board.forEachMove(map, [&moves] __host__ __device__ (Move<5> m) {
    moves.push_back(m);
    return CONTINUE;
  });

  std::cout << "Num moves: " << moves.size() << std::endl;
  for(int i = 0; i < 50; i++) {
    host_moves.insert(host_moves.end(), moves.begin(), moves.end());
  }

  int* host_score = new int[host_moves.size()];

  Board<5>* dev_board;
  Move<5>* dev_moves;
  int* dev_score;

  std::cout << "Num moves: " << host_moves.size() << std::endl;


  hipMalloc(&dev_board, sizeof(host_board));
  hipMalloc(&dev_moves, host_moves.size()*sizeof(host_moves[0]));
  hipMalloc(&dev_score, host_moves.size()*sizeof(host_score[0]));
  cudaCheckError();

  auto start = steady_clock::now();
  hipMemcpy(dev_board, &host_board, sizeof(host_board), hipMemcpyHostToDevice);
  hipMemcpy(dev_moves, host_moves.data(), host_moves.size()*sizeof(host_moves[0]), hipMemcpyHostToDevice);
  cudaCheckError();

  eval_parallel<<<NUM_BLOCKS,BLOCK_SIZE>>>(host_board.curPlayer, host_moves.size(), dev_board, dev_moves, dev_score);
  hipDeviceSynchronize();
  hipMemcpy(host_score, dev_score, host_moves.size()*sizeof(host_score[0]), hipMemcpyDeviceToHost);
  cudaCheckError();
  //for(auto s : host_score) {
    //host_score[0] = max(host_score[0], s);
  //}
  auto end = steady_clock::now();

  printf("parallel: %d, time: %lu\n", host_score[0], duration_cast<microseconds>(end-start).count());
  start = steady_clock::now();
  int i = 0;
  for(Move<5> move : host_moves) {
    Board<5> b = host_board;
    b.execute(move);
    host_score[i++] = Eval::eval(b,host_board.curPlayer);
  }
  end = steady_clock::now();
  printf("sequential, time: %lu\n", duration_cast<microseconds>(end-start).count());
}
